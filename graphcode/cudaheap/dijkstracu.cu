#include "hip/hip_runtime.h"
#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <iostream>
#include "graph.hpp"
#define ll long long

using namespace std;
//total size of the heap
#define maxSize 1000000000000

__global__ void Insert_Elem(volatile ll int *heap,ll int *d_elements,ll int *curSize,volatile ll int *lockArr,ll int *elemSize,ll int k){
    ll int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
    {
        ll  int childInd = atomicInc((long long unsigned *) curSize,maxSize);
        childInd = childInd*k;
        for(int i = 0;i<k;i++)
            heap[childInd+i] = d_elements[tid*k+i];

        ll  int parInd = ((childInd/k - 1)/2) * k;

        if(childInd == 0){
            lockArr[childInd] = 0;
        }

        if(childInd != 0)
        {
            int oldval = 1;
            do
            {
                oldval = atomicCAS((int*)&lockArr[parInd/k],0,1);
                if(oldval == 0) //if we got the lock on parent
                {
                    if(heap[parInd] > heap[childInd])
                    {
                        for(int i = 0;i<k;i++){
                            ll int temp = heap[parInd+i];    //swapping the elements
                            heap[parInd+i] = heap[childInd+i];
                            heap[childInd+i] = temp;
                           // __threadfence();//necessary
		                }
			            
                        __threadfence();

                        lockArr[childInd/k] = 0; //unlock the child
        
                        childInd = parInd;
                        parInd = ((childInd/k - 1)/2) * k;
                        oldval = 1; //we need to heapify again

                        //if we have reached the root
                        if(childInd == 0){
                            oldval = 0; //we need not heapify again
                            lockArr[childInd/k] = 0;
                        }  
                    }
                    else //if heap property satisfied release the locks
                    {
                        lockArr[childInd/k] = 0;
                        lockArr[parInd/k] = 0;
                    } 
                    
                }
                //__threadfence(); //doesnt seem necessary
            }while(oldval != 0);
        }
    }
}

bool checkHeap(ll int *ar,ll int size,ll int k)
{
    for(ll int i = 0;i<size/2;i+=k)
    {
        if(ar[i] > ar[2*i + k]){
            printf("\nproblem found at index parent = %lld,child = %lld\n",i,2*i + k);
            printf("problem found at index parentval = %lld,childval = %lld\n",ar[i],ar[2*i + k]); 
            return false;
        } 
        if((2*i + 2) < size && ar[i] > ar[2*i + 2*k]){
            printf("\nproblem found at index parent = %lld,child = %lld\n",i,2*i + 2*k);
            printf("problem found at index parentval = %lld,childval = %lld\n",ar[i],ar[2*i + 2*k]);
            return false;
        }
    }
    return true;
}

ll int getRandom(ll int lower,ll int upper)
{
    ll int num = (rand() % (upper - lower + 1)) + lower;
    return num;  
}
void printArray(ll int arr[],ll int size,ll int k)
{
    for(ll int i = 0;i<size;i++)
    {
        if(arr[i] == 10000000)
            printf("-1, ");
        else
            printf("%lld, ",arr[i]);
    }
    
    cout<<endl;
}
void FillArray(ll int elements[],ll int size,ll int k)
{
    for(ll int i = 0;i<size*k;i++)
    {
        elements[i] = getRandom(1,1000);
    }
}
    
void heapify(ll int hp[],ll int ind,ll int size,ll int k)
{
    while(1)
    {
        ll int leftChild = 2*ind+k;
        ll int rightChild = 2*ind+2*k;
        ll int largeInd = -1;
        if(rightChild < size*k && hp[ind] > hp[rightChild]){
            if(hp[leftChild] < hp[rightChild])
                largeInd = leftChild;
            else
                largeInd = rightChild;
        }
        else if(leftChild < size*k && hp[ind] > hp[leftChild]){
            largeInd = leftChild;
        }
        
        if(largeInd == -1)  return;
	
    
        for(ll int i = 0;i<k;i++){
            ll int temp = hp[ind+i];
            hp[ind+i] = hp[largeInd+i];
            hp[largeInd+i] = temp;
        }

        ind = largeInd;
        
    }

}

void heapifyBUP(ll int arr[], ll int n, ll int childInd, ll int k) {
    // Find parent 
    ll int parInd = ((childInd/k - 1)/2) * k;
    if (parInd >= 0) { 
        if (arr[childInd] < arr[parInd]) { 
            for(ll int i = 0;i<k;i++){
                ll int temp = arr[parInd+i];
                arr[parInd+i] = arr[childInd+i];
                arr[childInd+i] = temp;
            }
            heapifyBUP(arr, n, parInd,k); 
        } 
    } 
}

void insertNode(ll int arr[], ll  int& n, ll int Key,ll int val,ll int k)
{
    // Increase the size of Heap by 2
    n = n + 1;
    ll int childInd = n*k;
 
    // Insert the element at end of Heap
    arr[childInd - 2] = Key;
    arr[childInd - 1] = val;
 
    // Heapify the new node following a
    // Bottom-up approach
    heapifyBUP(arr, n,childInd-k,k);
}

void deleteRoot(ll int arr[],ll int *n,ll int k)
{
    for(ll int i = 0;i<k;i++){
        arr[i] = arr[(*n -1)*2 + i];
    }
 
    // Decrease size of heap by 1
    *n = *n - 1;
 
    // heapify the root node
    heapify(arr,0,*n,k);
}

void buildHeap(ll int hp[],ll int n,ll int k)
{
    for(ll int i = n/2 -1 ; i>=0;i--)
    {
        heapify(hp,i*k,n,k);
    }
}

__global__ void setLockVar(ll int *curSize,ll int *lockArr,ll int *elemSize)
{
    long long int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
        lockArr[tid + *curSize] = 1;
}

double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    ll int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) cout << stat;
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

void serDijkstra(ll int *pos,ll int *neigh,ll int *weight,ll int wn,ll int src,ll int dist[],ll int V,ll int k,double &serdijtime)
{
    double starttime = rtclock();

    long long int *hp;
    hipMalloc(&hp,(maxSize)*sizeof(ll int));
    ll int cursize = 0; 
    insertNode(hp, cursize,0,src,k);
    dist[src] = 0;
    ll int totDeg = 0,count = 0;

    while (cursize != 0) {
        ll int elem = hp[1];
        deleteRoot(hp,&cursize,k);
 
        ll int start = pos[elem];
        ll int end;
        if(elem+1 < V)
            end = pos[elem+1];
        else
            end = wn;

        if(end > start)
        {
            totDeg += (end-start);
            count++;
        }

        for(ll int i = start;i<end;i++)
        {
            ll int v = neigh[i];
            ll int wt = weight[i];
            if (dist[v] > dist[elem] + wt) {
                // Updating distance of v
                dist[v] = dist[elem] + wt ;
                insertNode(hp, cursize,dist[v],v,k);
            }
        }
    }
    double endtime = rtclock();  
    serdijtime = endtime - starttime;
    printtime("Serial Dijkstra Time Taken ", starttime, endtime);
    cout << "Avg degree : "<<totDeg/count;
}

void setGraph(graph &G,ll int *pos,ll int *neigh, ll int V, ll int E){

    int *temp = G.getEdgeLen();
    for(ll int i = 0; i < V; i++) {
        pos[i] = G.indexofNodes[i];
    }

    for(ll int i = 0; i < E; i++) {
        neigh[i] = G.edgeList[i];
    }

    // printArray(pos,pn,k);
    // printArray(neigh,wn,k);
    // printArray(weight,wn,k);

    cout <<"No of Verices : "<< V << endl;
    cout <<"No of Edges   : "<< E << endl;
}

void parDijkstra(ll  int *curSize, ll int *elemSize, ll int *d_elements, ll int *lockArr, ll int *d_a, ll int k,double &pardijtime, ll int dist[],
ll int V, ll int E, ll int *pos,ll int *neigh,ll int *weight, ll int Source){
    ll int totDeg = 0,count = 0;
    *curSize = 0;
    *elemSize = 0; //equal to the max degree in the graph

    // printArray(dist,V,k);
    double starttime = rtclock();  
    dist[Source] = 0;
    *elemSize = 1;
    d_elements[0] = 0;
    d_elements[1] = Source;

    //Initialization
    ll int block = ceil((float) *elemSize/1024);
    setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
    hipDeviceSynchronize();
    Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize,k);
    hipDeviceSynchronize();
    // cout << *curSize<<endl;
        
    while(*curSize != 0)
    {
        // printArray(d_a,*curSize*k,k);
        ll int elem = d_a[1];
        deleteRoot(d_a,curSize,k); //call delete here
        *elemSize = 0;
        // for(auto neigh : adj[elem])
        ll int start = pos[elem];
        ll int end;
        if(elem+1 < V)
            end = pos[elem+1];
        else
            end = E;

        // cout << "elem = "<<elem<<", start = "<<start<<", end = "<<end<<endl;
        if(end > start)
        {
            totDeg += (end-start);
            count++;
        }
        
        for(ll int i = start;i<end;i++)
        {
            ll int v = neigh[i];
            ll int wt = weight[i];
            
            // cout << "dist[elem] = "<<dist[elem]<<", wt = "<<wt<<", v = "<<v<<", dist[v] = "<<dist[v]<<endl;
            if(dist[elem] + wt < dist[v])
            {
                dist[v] = dist[elem]+wt;
                d_elements[*elemSize*2] = dist[v];
                d_elements[*elemSize*2 + 1] = v;
                *elemSize = *elemSize + 1;
            }
        }
        // printArray(d_elements,*elemSize*k,k);
        if(*elemSize != 0)
        {
            ll int block = ceil((float) *elemSize/1024);
            setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
            hipDeviceSynchronize();
            Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize,k);
            hipDeviceSynchronize();
        }
        // cout << *curSize<<endl;
        // printArray(d_a,*curSize*k,k);
        // printArray(dist,V,k);
        // cout<<"............................................."<<endl;
        
    }
    double endtime = rtclock();  
    pardijtime = endtime - starttime;
    printtime("Parallel Dijkstra Time Taken ", starttime, endtime);
    cout << "Avg degree : "<<totDeg/count;
}

void compareDis(ll int dist[],ll int dist2[],ll int V,ll int k)
{
    ll int flag = 0;
    for(ll int i = 0;i<V;i++){
        if(dist[i] != dist2[i]){
            flag++;
	    if(flag < 100)
	    cout << i << ", "<<dist[i]<<", "<<dist2[i]<<endl;
        }
    }

    cout << "No of Errors : "<<flag << endl;

    //printArray(dist,V,k);
    //printArray(dist2,V,k);
}

int main(int argc, char* argv[]) {

    graph G(argv[1]);
    G.parseGraph();
    srand(time(0));

    ll int *curSize;
    ll int *d_a,*lockArr,*elemSize,Source = 0,*d_elements,k = 2,*pos,*neigh,*weight;
    ll int V = G.num_nodes();
    ll int E = G.num_edges();
    hipHostAlloc(&curSize, sizeof(ll int), 0);
    hipHostAlloc(&elemSize, sizeof(ll int), 0);
    hipMalloc(&lockArr,(V+10)*sizeof(ll int));
    hipMalloc(&pos,(V+10)*sizeof(ll int));
    hipMalloc(&neigh,(E+10)*sizeof(ll int));
    hipMalloc(&weight,(E+10)*sizeof(ll int));
    hipMemset(lockArr,0,(V+10)*sizeof(ll int));
    hipHostAlloc(&d_a, maxSize*k*sizeof(ll int),0);
    hipHostAlloc(&d_elements, maxSize * sizeof(ll int),0);

    //int V = G.num_nodes();
    
    int *temp = G.getEdgeLen();

    for(ll int i = 0; i < E; i++) {
        weight[i] = temp[i];
    }

    setGraph(G,pos,neigh,V,E);

    ll int *dist,*dist2;
    hipMalloc(&dist,(V+10)*sizeof(ll int));
    hipMalloc(&dist2,(V+10)*sizeof(ll int));

    for(ll int i = 0;i<V;i++) {
        dist[i] =  1000000000;
        dist2[i] = 1000000000;
    }

    double serdijtime,pardijtime;
    parDijkstra(curSize,elemSize,d_elements,lockArr,d_a,k,pardijtime,dist,V,E,pos,neigh,weight,Source);
    serDijkstra(pos,neigh,weight,E,Source,dist2,V,k,serdijtime);
    
    double perf = serdijtime/pardijtime;
    
    cout << "Performance Gained :"<<perf<<endl;

    compareDis(dist,dist2,V,k);

    return 0;
}
