// FOR BC: nvcc bc_dsl_v2.cu -arch=sm_60 -std=c++14 -rdc=true # HW must support CC 6.0+ Pascal or after
// This code is incomplete. Work is going on in this.
#include "DynamicSCC_cuda.h"

void staticSCC(graph &g, int* d_meta, int* d_data, int* d_rev_meta, int* d_src, bool* h_isPivot, int* h_range, int* scc, bool* d_isPivot, int* d_range, int* d_scc)
{
  int V = g.num_nodes();
  int E = g.num_edges();

  printf("#nodes:%d\n",V);
  printf("#edges:%d\n",E);
  int* edgeLen = g.getEdgeLen();

  //LAUNCH CONFIG
  const unsigned threadsPerBlock = 512;
  unsigned numThreads   = (V < threadsPerBlock)? 512: V;
  unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;


  // TIMER START
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);


  //DECLAR DEVICE AND HOST vars in params

  //BEGIN DSL PARSING 
  bool* d_modified;
  hipMalloc(&d_modified, sizeof(bool)*(V));

  bool* d_modified_next;
  hipMalloc(&d_modified_next, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_modified,(bool)false);

  int* d_outDeg;
  hipMalloc(&d_outDeg, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_outDeg,(int)0);

  int* d_inDeg;
  hipMalloc(&d_inDeg, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_inDeg,(int)0);

  bool* d_visitFw;
  hipMalloc(&d_visitFw, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitFw,(bool)false);

  bool* d_visitBw;
  hipMalloc(&d_visitBw, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_visitBw,(bool)false);

  bool* d_propFw;
  hipMalloc(&d_propFw, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propFw,(bool)false);

  bool* d_propBw;
  hipMalloc(&d_propBw, sizeof(bool)*(V));

  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V,d_propBw,(bool)false);

  int* d_pivotField;
  hipMalloc(&d_pivotField, sizeof(int)*(V));

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_pivotField,(int)-1);

  vHong_kernel1<<<numBlocks, threadsPerBlock>>>(V,E,d_outDeg,d_inDeg,d_meta,d_rev_meta);
  hipDeviceSynchronize();



  ; // asst in .cu

  bool fpoint1 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  int k=0; // #fixpt-Iterations
  while(!fpoint1) {

    fpoint1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel2<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_range,d_scc,d_isPivot);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT

  vHong_kernel3<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_outDeg,d_inDeg,d_pivotField,d_range);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  ; // asst in .cu

  vHong_kernel4<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_pivotField,d_range,d_isPivot,d_visitBw,d_visitFw);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  bool fpoint2 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  k=0; // #fixpt-Iterations
  while(!fpoint2) {

    fpoint2 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel5<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_propBw,d_visitBw,d_propFw,d_visitFw,d_scc,d_range);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT

  vHong_kernel6<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_visitFw,d_visitBw,d_range);
  hipDeviceSynchronize();



  ; // asst in .cu

  ; // asst in .cu

  vHong_kernel7<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_visitFw,d_visitBw,d_propBw,d_propFw);
  hipDeviceSynchronize();



  fpoint1 = false;
  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  k=0; // #fixpt-Iterations
  while(!fpoint1) {

    fpoint1 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint1), &fpoint1, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel8<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_range,d_scc,d_isPivot);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu

    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint1, HIP_SYMBOL(::fpoint1), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT

  initKernel<int> <<<numBlocks,threadsPerBlock>>>(V,d_range,(int)0);

  vHong_kernel9<<<numBlocks, threadsPerBlock>>>(V,E,d_range);
  hipDeviceSynchronize();



  bool fpoint4 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  k=0; // #fixpt-Iterations
  while(!fpoint4) {

    fpoint4 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint4), &fpoint4, sizeof(bool), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint4), &fpoint4, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel10<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_range,d_scc);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint4, HIP_SYMBOL(::fpoint4), sizeof(bool), 0, hipMemcpyDeviceToHost);



    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint4), &fpoint4, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel11<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_range);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint4, HIP_SYMBOL(::fpoint4), sizeof(bool), 0, hipMemcpyDeviceToHost);



    ; // asst in .cu


    hipMemcpyFromSymbol(&fpoint4, HIP_SYMBOL(::fpoint4), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT

  bool fpoint5 = false; // asst in .cu

  // FIXED POINT variables
  //BEGIN FIXED POINT
  initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
  k=0; // #fixpt-Iterations
  while(!fpoint5) {

    fpoint5 = true;
    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint5), &fpoint5, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel12<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_outDeg,d_inDeg,d_pivotField,d_range);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    ; // asst in .cu

    vHong_kernel13<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_pivotField,d_range,d_isPivot,d_visitBw,d_visitFw);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    fpoint2 = false;
    // FIXED POINT variables
    //BEGIN FIXED POINT
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k=0; // #fixpt-Iterations
    while(!fpoint2) {

      fpoint2 = true;
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
      hipMemcpyToSymbol(HIP_SYMBOL(::fpoint2), &fpoint2, sizeof(bool), 0, hipMemcpyHostToDevice);
      vHong_kernel14<<<numBlocks, threadsPerBlock>>>(V,E,d_meta,d_data,d_src,d_rev_meta,d_propBw,d_visitBw,d_propFw,d_visitFw,d_scc,d_range);
      hipDeviceSynchronize();
      hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);



      ; // asst in .cu


      hipMemcpyFromSymbol(&fpoint2, HIP_SYMBOL(::fpoint2), sizeof(bool), 0, hipMemcpyDeviceToHost);
      hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
      initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
      k++;
    } // END FIXED POINT

    vHong_kernel15<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_visitFw,d_visitBw,d_range);
    hipDeviceSynchronize();



    ; // asst in .cu

    ; // asst in .cu

    hipMemcpyToSymbol(HIP_SYMBOL(::fpoint5), &fpoint5, sizeof(bool), 0, hipMemcpyHostToDevice);
    vHong_kernel16<<<numBlocks, threadsPerBlock>>>(V,E,d_scc,d_visitFw,d_visitBw,d_propBw,d_propFw);
    hipDeviceSynchronize();
    hipMemcpyFromSymbol(&fpoint5, HIP_SYMBOL(::fpoint5), sizeof(bool), 0, hipMemcpyDeviceToHost);




    hipMemcpyFromSymbol(&fpoint5, HIP_SYMBOL(::fpoint5), sizeof(bool), 0, hipMemcpyDeviceToHost);
    hipMemcpy(d_modified, d_modified_next, sizeof(bool)*V, hipMemcpyDeviceToDevice);
    initKernel<bool> <<<numBlocks,threadsPerBlock>>>(V, d_modified_next, false);
    k++;
  } // END FIXED POINT

  hipMemcpy(h_isPivot, d_isPivot, sizeof(bool)*V, hipMemcpyDeviceToHost);
  hipMemcpy(h_range, d_range, sizeof(int)*V, hipMemcpyDeviceToHost);
  hipMemcpy(scc, d_scc, sizeof(int)*V, hipMemcpyDeviceToHost);

  //hipFree up!! all propVars in this BLOCK!
  hipFree(d_propBw);
  hipFree(d_propFw);
  hipFree(d_visitBw);
  hipFree(d_visitFw);
  hipFree(d_inDeg);
  hipFree(d_outDeg);
  hipFree(d_pivotField);
  hipFree(d_modified);

  //TIMER STOP
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("GPU Time: %.6f ms\n", milliseconds);

} //end FUN
void DynSCC(graph &g, std::vector<update> updateBatch, int batchSize) {
	int V = g.num_nodes();
	int E = g.num_edges();
	
	int *h_meta;
	int *h_data;
	int *h_src;
	int *h_rev_meta;

	h_meta = (int *)malloc( (V+1)*sizeof(int));
	h_data = (int *)malloc( (E)*sizeof(int));
	h_src = (int *)malloc( (E)*sizeof(int));
	h_rev_meta = (int *)malloc( (V+1)*sizeof(int));

	for(int i=0; i<= V; i++) {
		int temp;
		temp = g.indexofNodes[i];
		h_meta[i] = temp;
		temp = g.rev_indexofNodes[i];
		h_rev_meta[i] = temp;
	}

	for(int i=0; i< E; i++) {
		int temp;
		temp = g.edgeList[i];
		h_data[i] = temp;
		temp = g.srcList[i];
		h_src[i] = temp;
	}

	int* d_meta;
	int* d_data;
	int* d_src;
	int* d_rev_meta;

	hipMalloc(&d_meta, sizeof(int)*(1+V));
	hipMalloc(&d_data, sizeof(int)*(E));
	hipMalloc(&d_src, sizeof(int)*(E));
	hipMalloc(&d_rev_meta, sizeof(int)*(V+1));

	hipMemcpy(  d_meta,   h_meta, sizeof(int)*(V+1), hipMemcpyHostToDevice);
	hipMemcpy(  d_data,   h_data, sizeof(int)*(E), hipMemcpyHostToDevice);
	hipMemcpy(   d_src,    h_src, sizeof(int)*(E), hipMemcpyHostToDevice);
	hipMemcpy(d_rev_meta, h_rev_meta, sizeof(int)*((V+1)), hipMemcpyHostToDevice);
	
	const unsigned threadsPerBlock = 512;
	unsigned numThreads   = (V < threadsPerBlock)? 512: V;
	unsigned numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
	
	bool* isPivot = (bool*)malloc(V*sizeof(bool));
	int* range = (int*)malloc(V*sizeof(int));
	int* scc = (int*)malloc(V*sizeof(int));

	bool* d_isPivot;
	int* d_range;
	int* d_scc;

	hipMalloc(&d_isPivot, V*sizeof(bool));
	hipMalloc(&d_range, V*sizeof(int));
	hipMalloc(&d_scc, V*sizeof(int));

	staticSCC(g, d_meta, d_data, d_rev_meta, d_src, isPivot, range, scc, d_isPivot, d_range, d_scc);

	int vertices = 0;
	for (int i = 0; i < V; i++){
		if (isPivot[i]){
			vertices++;
		}
	}
	
	int *condense_meta = (int *)malloc(vertices * sizeof(int));
	int *condense_edge = (int *)malloc(E * sizeof(int));
	int *mapper = (int *)malloc(2 * vertices * sizeof(int));
	int k = 0;
	for (int i = 0; i < V; i++) {
		if (isPivot[i]) {
			mapper[k] = i;
			k += 2;
		}
	}
	k = 0;
	for (int i = 1; i < 2 * vertices; i += 2){
		mapper[i] = k;
		k++;
	}
	
	int* d_source;
	int* d_destination;
	int* d_mapper;
	int* d_pivot;
	int* d_edges;
	int* d_mark;
	int* locks;
	int* parents;
	int* d_k;
	int* d_mapper_vertices;
	
	hipMalloc(&d_source, E * sizeof(int));
	hipMalloc(&d_destination, E * sizeof(int));
	hipMalloc(&d_mapper, 2 * vertices * sizeof(int));
	hipMalloc(&d_pivot, V*sizeof(int));
	hipMalloc(&d_edges, E*sizeof(int));
	hipMalloc(&locks, E*sizeof(int));
	hipMalloc(&parents, vertices*sizeof(int));
	hipMalloc(&d_k, sizeof(int));
	hipMalloc(&d_mapper_vertices, E*sizeof(int));
	
	initKernel<int> <<<numBlocks,numThreads>>>(V,d_range,(int)0);
	initKernel<int> <<<numBlocks,numThreads>>>(V,d_scc,(int)-1);
	initKernel<bool> <<<numBlocks,numThreads>>>(V,d_isPivot,(bool)false);
	initKernel<int> <<<numBlocks,numThreads>>>(E,d_edges,(int)-1);
	initKernel<int> <<<numBlocks,numThreads>>>(E,d_source,(int)-1);
	initKernel<int> <<<numBlocks,numThreads>>>(E,d_destination,(int)-1);
	
	numThreads   = (V < threadsPerBlock)? 512: V;
	numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
	
	hipMemcpy(range, d_range, V*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(d_mapper, mapper, 2*vertices*sizeof(int), hipMemcpyHostToDevice);
	
	int batchElements = 0;
	int* mark = (int*)malloc(2 * batchSize * sizeof(int));
	hipMalloc(&d_mark, 2 * batchSize * sizeof(int));
	for (int updateIndex = 0; updateIndex < updateBatch.size(); updateIndex += batchSize) {
		if ((updateIndex + batchSize) > updateBatch.size()) {
				batchElements = updateBatch.size() - updateIndex;
		}
		else
			batchElements = batchSize;
		
		hipMemset(d_k, 0, sizeof(int));
		
		int index = 0;
		set_pivot<<<numBlocks, numThreads>>>(V, d_pivot, d_isPivot, d_range);
		create_graph<<<numBlocks, numThreads>>>(V, E, vertices, d_meta, d_data, d_source, d_destination, d_mapper, d_edges, d_isPivot, d_range, d_k, d_pivot);
		
		bool fpoint = false;
		while (!fpoint)
		{
			hipMemcpyToSymbol(HIP_SYMBOL(::fpoint), &fpoint, sizeof(bool), 0, hipMemcpyHostToDevice);
			clean_graph<<<numBlocks, numThreads>>>(E, d_source, d_destination);
			hipMemcpyFromSymbol(&fpoint, HIP_SYMBOL(::fpoint), sizeof(bool), 0, hipMemcpyDeviceToHost);
		}
		
		index = 0;
		for (int batchIndex = updateIndex; batchIndex < (updateIndex + batchSize) && batchIndex < updateBatch.size(); batchIndex++) {
			if (updateBatch[batchIndex].type == 'a') {
				update u = updateBatch[batchIndex];
				int src = u.source;
				int dst = u.destination;
				mark[index++] = src;
				mark[index++] = dst;
			}
		}
		g.updateCSRAdd(updateBatch, updateIndex, batchElements);
		
		numThreads   = (index / 2 < threadsPerBlock)? 512: index / 2;
		numBlocks    = (index / 2 + threadsPerBlock-1)/threadsPerBlock;
		
		initKernel<int> <<<numBlocks,numThreads>>>(E,locks,(int)0);
		initKernel<int> <<<numBlocks,numThreads>>>(vertices,parents,(int)-1);
		
		numThreads   = (E < threadsPerBlock)? 512: E;
		numBlocks    = (E+threadsPerBlock-1)/threadsPerBlock;
		
		initKernel<int> <<<numBlocks,threadsPerBlock>>>(E,d_edges,(int)-1);
		
		hipMemcpy(d_mark, mark, 2 * batchSize * sizeof(int), hipMemcpyHostToDevice);
		update_condense_graph<<<numBlocks, numThreads>>>(vertices, E, d_source, d_destination, d_k, d_mark, d_pivot, d_mapper);
		check_cycle<<<numBlocks,numThreads>>>(vertices, E, d_source, d_destination, d_k, d_mark, d_mapper, d_pivot, d_edges, locks, parents, d_mapper_vertices);
		check_cycle_repeat<<<numBlocks,numThreads>>>(vertices, E, d_source, d_destination, d_k, d_mark, d_mapper, d_pivot, d_edges, locks, parents, d_mapper_vertices);
		
		fpoint = false;
		while (!fpoint)
		{
			hipMemcpyToSymbol(HIP_SYMBOL(::fpoint), &fpoint, sizeof(bool), 0, hipMemcpyHostToDevice);
			clean_graph<<<numBlocks, numThreads>>>(E, d_source, d_destination);
			hipMemcpyFromSymbol(&fpoint, HIP_SYMBOL(::fpoint), sizeof(bool), 0, hipMemcpyDeviceToHost);
		}
		
		numThreads   = (V < threadsPerBlock)? 512: V;
		numBlocks    = (V+threadsPerBlock-1)/threadsPerBlock;
		
		graph_recolour<<<numBlocks, numThreads>>>(V, d_range, d_pivot);
	}
	
	hipFree(d_meta);
	hipFree(d_data);
	hipFree(d_rev_meta);
	hipFree(d_src);
	hipFree(d_mapper);
	hipFree(d_pivot);
	hipFree(d_isPivot);
	hipFree(d_range);
	hipFree(d_scc);
	hipFree(locks);
	hipFree(parents);
	hipFree(d_source);
	hipFree(d_destination);
	hipFree(d_edges);
	hipFree(d_mark);
	hipFree(d_k);
	
	int count_scc = 0;
	for (int i = 0; i < V; i++) {
		if (d_isPivot[i]) {
			count_scc++;
		}
	}
}

/*int main(int argc, char*argv[])
{
	std::string s(argv[1]);
	double start, end;
	if (argc > 1)
		;
	else
	{
		std::cout << "Give the path to the input graph text file" << std::endl;
		std::cin >> s;
	}
	char *c = new char[s.length() + 1];
	std::copy(s.begin(), s.end(), c);
	graph g(c);
	g.parseGraph();
	DynSCC(g);
	return 0;
}*/
