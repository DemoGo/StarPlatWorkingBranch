#include "hip/hip_runtime.h"
#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <fstream>
#include <time.h>
#include <iostream>
#include "../graph.hpp"

using namespace std;
//total size of the heap
#define maxSize 1000000

__global__ void Insert_Elem(volatile int *heap,int *d_elements,int *curSize,volatile int *lockArr,int *elemSize,int k){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
    {
        int childInd = atomicInc((unsigned *) curSize,maxSize+10);
        childInd = childInd*k;
        for(int i = 0;i<k;i++)
            heap[childInd+i] = d_elements[tid*k+i];

        int parInd = ((childInd/k - 1)/2) * k;

        if(childInd == 0){
            lockArr[childInd] = 0;
        }

        if(childInd != 0)
        {
            int oldval = 1;
            do
            {
                oldval = atomicCAS((int*)&lockArr[parInd/k],0,1);
                if(oldval == 0) //if we got the lock on parent
                {
                    if(heap[parInd] > heap[childInd])
                    {
                        for(int i = 0;i<k;i++){
                            int temp = heap[parInd+i];    //swapping the elements
                            heap[parInd+i] = heap[childInd+i];
                            heap[childInd+i] = temp;
		                }

                        __threadfence();//necessary

                        lockArr[childInd/k] = 0; //unlock the child
        
                        childInd = parInd;
                        parInd = ((childInd/k - 1)/2) * k;
                        oldval = 1; //we need to heapify again

                        //if we have reached the root
                        if(childInd == 0){
                            oldval = 0; //we need not heapify again
                            lockArr[childInd/k] = 0;
                        }  
                    }
                    else //if heap property satisfied release the locks
                    {
                        lockArr[childInd/k] = 0;
                        lockArr[parInd/k] = 0;
                    } 
                    
                }
                // __threadfence(); //doesnt seem necessary
            }while(oldval != 0);
        }
    }
}

bool checkHeap(int *ar,int size,int k)
{
    for(int i = 0;i<size/2;i+=k)
    {
        if(ar[i] > ar[2*i + k]){
            printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + k);
            printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + k]); 
            return false;
        } 
        if((2*i + 2) < size && ar[i] > ar[2*i + 2*k]){
            printf("\nproblem found at index parent = %d,child = %d\n",i,2*i + 2*k);
            printf("problem found at index parentval = %d,childval = %d\n",ar[i],ar[2*i + 2*k]);
            return false;
        }
    }
    return true;
}

int getRandom(int lower, int upper)
{
    int num = (rand() % (upper - lower + 1)) + lower;
    return num;  
}
void printArray(int arr[],int size,int k)
{
    for(int i = 0;i<size;i++)
    {
        if(arr[i] == 10000000)
            printf("-1, ");
        else
            printf("%d, ",arr[i]);
    }
    
    cout<<endl;
}
void FillArray(int elements[],int size,int k)
{
    for(int i = 0;i<size*k;i++)
    {
        elements[i] = getRandom(1,1000);
    }
}
    
void heapify(int hp[],int ind,int size,int k)
{
    while(1)
    {
        int leftChild = 2*ind+k;
        int rightChild = 2*ind+2*k;
        int largeInd = -1;
        if(rightChild < size*k && hp[ind] > hp[rightChild]){
            if(hp[leftChild] < hp[rightChild])
                largeInd = leftChild;
            else
                largeInd = rightChild;
        }
        else if(leftChild < size*k && hp[ind] > hp[leftChild]){
            largeInd = leftChild;
        }
        
        if(largeInd == -1)  return;
	
    
        for(int i = 0;i<k;i++){
            int temp = hp[ind+i];
                hp[ind+i] = hp[largeInd+i];
                hp[largeInd+i] = temp;
        }

        ind = largeInd;
        
    }

}

void heapifyBUP(int arr[], int n, int childInd,int k) {
    // Find parent 
    int parInd = ((childInd/k - 1)/2) * k;
    if (parInd >= 0) { 
        if (arr[childInd] < arr[parInd]) { 
            for(int i = 0;i<k;i++){
                int temp = arr[parInd+i];
                arr[parInd+i] = arr[childInd+i];
                arr[childInd+i] = temp;
            }
            heapifyBUP(arr, n, parInd,k); 
        } 
    } 
}

void insertNode(int arr[], int& n, int Key,int val,int k)
{
    // Increase the size of Heap by 2
    n = n + 1;
    int childInd = n*k;
 
    // Insert the element at end of Heap
    arr[childInd - 2] = Key;
    arr[childInd - 1] = val;
 
    // Heapify the new node following a
    // Bottom-up approach
    heapifyBUP(arr, n,childInd-k,k);
}

void deleteRoot(int arr[], int *n,int k)
{
    for(int i = 0;i<k;i++){
        arr[i] = arr[(*n -1)*2 + i];
    }
 
    // Decrease size of heap by 1
    *n = *n - 1;
 
    // heapify the root node
    heapify(arr,0,*n,k);
}

void buildHeap(int hp[],int n,int k)
{
    for(int i = n/2 -1 ; i>=0;i--)
    {
        heapify(hp,i*k,n,k);
    }
}

__global__ void setLockVar(int *curSize,int *lockArr,int *elemSize)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < *elemSize)
        lockArr[tid + *curSize] = 1;
}

double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

void serDijkstra(int pos[],int neigh[],int weight[],int wn,int src,int dist[],int V,int k,double &serdijtime)
{
    double starttime = rtclock();
    int hp[maxSize];
    int cursize = 0; 
    insertNode(hp, cursize,0,src,k);
    dist[src] = 0;
 

    while (cursize != 0) {
        int elem = hp[1];
        deleteRoot(hp,&cursize,k);
 
        int start = pos[elem];
        int end;
        if(elem+1 < V)
            end = pos[elem+1];
        else
            end = wn;

        for(int i = start;i<end;i++)
        {
            int v = neigh[i];
            int wt = weight[i];
            if (dist[v] > dist[elem] + wt) {
                // Updating distance of v
                dist[v] = dist[elem] + wt ;
                insertNode(hp, cursize,dist[v],v,k);
            }
        }
    }
    double endtime = rtclock();  
    serdijtime = endtime - starttime;
    printtime("Serial Dijkstra Time Taken ", starttime, endtime);
}

void setGraph(graph &G,int pos[],int neigh[],int V,int E){

    for(int i = 0; i < V; i++) {
        pos[i] = G.indexofNodes[i];
    }

    for(int i = 0; i < E; i++) {
        neigh[i] = G.edgeList[i];
    }

    // printArray(pos,pn,k);
    // printArray(neigh,wn,k);
    // printArray(weight,wn,k);

    cout <<"No of Verices : "<< V << endl;
    cout <<"No of Edges   : "<< E << endl;
}

void parDijkstra(int *curSize, int *elemSize,int *d_elements,int *lockArr,int *d_a,int k,double &pardijtime,int dist[],
int V,int E,int pos[],int neigh[],int weight[],int Source){
    *curSize = 0;
    *elemSize = 0; //equal to the max degree in the graph

    // printArray(dist,V,k);
    double starttime = rtclock();  
    dist[Source] = 0;
    *elemSize = 1;
    d_elements[0] = 0;
    d_elements[1] = Source;

    //Initialization
    int block = ceil((float) *elemSize/1024);
    setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
    hipDeviceSynchronize();
    Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize,k);
    hipDeviceSynchronize();
    // cout << *curSize<<endl;
        
    while(*curSize != 0)
    {
        // printArray(d_a,*curSize*k,k);
        int elem = d_a[1];
        deleteRoot(d_a,curSize,k); //call delete here
        *elemSize = 0;
        // for(auto neigh : adj[elem])
        int start = pos[elem];
        int end;
        if(elem+1 < V)
            end = pos[elem+1];
        else
            end = E;

        // cout << "elem = "<<elem<<", start = "<<start<<", end = "<<end<<endl;
        for(int i = start;i<end;i++)
        {
            int v = neigh[i];
            int wt = weight[i];
            // cout << "dist[elem] = "<<dist[elem]<<", wt = "<<wt<<", v = "<<v<<", dist[v] = "<<dist[v]<<endl;
            if(dist[elem] + wt < dist[v])
            {
                dist[v] = dist[elem]+wt;
                d_elements[*elemSize*2] = dist[v];
                d_elements[*elemSize*2 + 1] = v;
                *elemSize = *elemSize + 1;
            }
        }
        // printArray(d_elements,*elemSize*k,k);
        if(*elemSize != 0)
        {
            int block = ceil((float) *elemSize/1024);
            setLockVar<<<block,1024>>>(curSize,lockArr,elemSize);
            hipDeviceSynchronize();
            Insert_Elem<<<block,1024>>>(d_a,d_elements,curSize,lockArr,elemSize,k);
            hipDeviceSynchronize();
        }
        // cout << *curSize<<endl;
        // printArray(d_a,*curSize*k,k);
        // printArray(dist,V,k);
        // cout<<"............................................."<<endl;
        
    }
    double endtime = rtclock();  
    pardijtime = endtime - starttime;
    printtime("Parallel Dijkstra Time Taken ", starttime, endtime);
}

void compareDis(int dist[],int dist2[],int V,int k)
{
    int flag = 0;
    for(int i = 0;i<V;i++){
        if(dist[i] != dist2[i]){
            flag++;
        }
    }

    cout << "No of Errors : "<<flag << endl;

    printArray(dist,V,k);
    printArray(dist2,V,k);
}

int main(int argc, char* argv[]) {

    graph G(argv[1]);
	G.parseGraph();
    srand(time(0));

    int *d_a,*curSize,*lockArr,*elemSize,Source = 0,*d_elements,k = 2;

    hipHostAlloc(&curSize, sizeof(int), 0);
    hipHostAlloc(&elemSize, sizeof(int), 0);
    hipMalloc(&lockArr,(maxSize)*sizeof(int));
    hipMemset(lockArr,0,(maxSize)*sizeof(int));
    hipHostAlloc(&d_a, maxSize*k*sizeof(int),0);
    hipHostAlloc(&d_elements, maxSize * sizeof(int),0);

    int V = G.num_nodes();
    int E = G.num_edges();

    int pos[V],neigh[E],*weight = G.getEdgeLen();;
    setGraph(G,pos,neigh,V,E);
    int dist[V],dist2[V];

    for(int i = 0;i<V;i++) {
        dist[i] = 10000000;
        dist2[i] = 10000000;
    }

    double serdijtime,pardijtime;
    parDijkstra(curSize,elemSize,d_elements,lockArr,d_a,k,pardijtime,dist,V,E,pos,neigh,weight,Source);
    serDijkstra(pos,neigh,weight,E,Source,dist2,V,k,serdijtime);
    
    double perf = serdijtime/pardijtime;
    
    cout << "Performance Gained :"<<perf<<endl;

    compareDis(dist,dist2,V,k);

    return 0;
}
